#include "hip/hip_runtime.h"
#include "common.h"




int main(int argc, char **argv)
{
	
	// printf("sizeof(ShadeRec): %i\n",sizeof(ShadeRec));
	// printf("sizeof(Ray): %i\n",sizeof(Ray));
	//printf("sizeof(World): %i\n",sizeof(World));
	// printf("sizeof(float3): %i\n",sizeof(float3));
	//printf("sizeof(Compound): %i\n",sizeof(Compound));
	// printf("sizeof(GeometricObject): %i\n",sizeof(GeometricObject));
	//printf("sizeof(Grid): %i\n",sizeof(Grid));
	//printf("sizeof(Material): %i\n",sizeof(Material));
	
	// Material *mat = new Material();
	// uint mat_id = (uint)mat;
	// printf("0x%x, 0x%x\n",mat,mat_id);
	
	// exit(0);
	
	World world;
	if (!world.processArgs(argc,argv))
	{
		exit(0);
	}
	
	//CUT_DEVICE_INIT(argc,argv);  
	// world.ropt.useGrid = true;
	world.render();

	return 0;
}



